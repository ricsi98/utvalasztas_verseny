
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
__device__ int numOfDistance(int n, int d) {
    const int diameter = (n-1) / 2;
    if (d == 0) {
        return 1;
    }
    if (d <= diameter) {
        return 4 * d;
    }
    else {
        int z = 2*diameter - d + 1;
        return 4 * z;
    }
}

__device__ int2 distanceIndex2Offset(int distance, int index, int n) {
    const int num = numOfDistance(n*2+1, distance);
    const int fieldsPerSide = num / 4;
    const int fieldIdx = index / fieldsPerSide;
    const int remainder = index % fieldsPerSide;

    const int minX = max(1, distance - n);
    const int dx = minX + remainder;
    const int dy = distance - dx;

    if (fieldIdx == 0) {
        return make_int2(dx, dy);
    }
    if (fieldIdx == 1) {
        return make_int2(dy, -dx);
    }
    if (fieldIdx == 2) {
        return make_int2(-dx, -dy);
    }
    if (fieldIdx  == 3) {
        return make_int2(-dy, dx);
    }
}

__device__ int sampleAlias(hiprandState* state, int* alias, float* threshold, int n) {
    int offset = hiprand(state) % n;
    float alpha = hiprand_uniform(state);

    if (threshold[offset] < alpha) {
        return alias[offset];
    } else {
        return offset;
    }
}

__device__ int2 sampleOffsetNaive(hiprandState* state, int* alias, float* threshold, int n) {
    const int distance = sampleAlias(state, alias, threshold, n);
    const int num = numOfDistance(n, distance);
    const int index = hiprand(state) % num;
    return distanceIndex2Offset(distance, index, n);
}

__device__ int2 sampleOffset(hiprandState* state, int2 current, int* alias, float* threshold, int n) {
    while (true) {
        int2 offset = sampleOffsetNaive(state, alias, threshold, n);
        int x = offset.x + current.x;
        int y = offset.y + current.y;
        if (0 <= x && x < n && 0 <= y && y < n) {
            return make_int2(x,y);
        }
    }
}

extern "C" __global__
void my_fn(int* alias, float* threshold, int* y, int n, int cx, int cy) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init(0, tid, 0, &state);
    int2 current = make_int2(cx, cy);
    int2 offset = sampleOffset(&state, current, alias, threshold, n);
    y[tid*2] = offset.x;
    y[tid*2+1] = offset.y;
}